#include "hip/hip_runtime.h"
/*
//
//  Copyright 2010 SRI International
//
//  This file is part of the Computational Morphometry Toolkit.
//
//  http://www.nitrc.org/projects/cmtk/
//
//  The Computational Morphometry Toolkit is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  The Computational Morphometry Toolkit is distributed in the hope that it
//  will be useful, but WITHOUT ANY WARRANTY; without even the implied
//  warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with the Computational Morphometry Toolkit.  If not, see
//  <http://www.gnu.org/licenses/>.
//
//  $Revision$
//
//  $LastChangedDate$
//
//  $LastChangedBy$
//
*/

#include "cmtkDeviceHistogram_kernels.h"

__global__
void 
cmtkDeviceHistogramPLogPKernel( float* result, const float *dataPtr )
{
  int tx = threadIdx.x;

  // first, load data into shared memory
  __shared__ float working[512]; // allocate maximum we possibly need
  working[tx] = dataPtr[tx];
  __syncthreads();

  // second, compute sum of all bin values via butterfly
  for ( int bit = 1; bit <= blockDim.x; bit <<= 1 )
    {
      working[tx] += working[tx^bit];
      __syncthreads();
    }
  
  // third, normalize
  working[tx] = dataPtr[tx] / working[tx];
  __syncthreads();

  // fourth, do p*log(p)
  working[tx] *= log( working[tx] );
  __syncthreads();

  // fifth, another butterfly to compute \sum[p*log(p)]
  for ( int bit = 1; bit <= blockDim.x; bit <<= 1 )
    {
      working[tx] += working[tx^bit];
      __syncthreads();
    }

  result[tx] = working[tx];
}

void
cmtkDeviceHistogramPLogP( float* result, const float* dataPtr, int numberOfBins )
{
  dim3 dimBlock( numberOfBins, 1 );
  dim3 dimGrid( 1, 1 );
  
  cmtkDeviceHistogramPLogPKernel<<<dimGrid,dimBlock>>>( result, dataPtr );
}
