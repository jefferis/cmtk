#include "hip/hip_runtime.h"
/*
//
//  Copyright 2010 SRI International
//
//  This file is part of the Computational Morphometry Toolkit.
//
//  http://www.nitrc.org/projects/cmtk/
//
//  The Computational Morphometry Toolkit is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  The Computational Morphometry Toolkit is distributed in the hope that it
//  will be useful, but WITHOUT ANY WARRANTY; without even the implied
//  warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with the Computational Morphometry Toolkit.  If not, see
//  <http://www.gnu.org/licenses/>.
//
//  $Revision$
//
//  $LastChangedDate$
//
//  $LastChangedBy$
//
*/

#include "cmtkImageSymmetryPlaneFunctionalDevice_kernels.h"

#include <hip/hip_runtime_api.h>

#include <cstdio>

/// Texture reference to volume data.
texture<float, 3, hipReadModeElementType> texRef;
texture<float, 3, hipReadModeElementType> texRefX;

__global__
void cmtkImageSymmetryPlaneFunctionalDeviceEvaluateKernel( const float matrix[4][4], const float delta[3], const int dims0, const int dims1, const int dims2 )
{
  const int tx = threadIdx.x;

  __shared__ float sq[32];
  sq[tx] = 0;

  const int y = threadIdx.y;
  const int z = blockIdx.x;

  const float Y = y * delta[1];
  const float Z = z * delta[2];

  const float mXo = Y * matrix[1][0] + Z * matrix[2][0] + matrix[3][0];
  const float mYo = Y * matrix[1][1] + Z * matrix[2][1] + matrix[3][1];
  const float mZo = Y * matrix[1][2] + Z * matrix[2][2] + matrix[3][2];

  for ( int x = tx; x < dims0; x += blockDim.x )
    {
      const float X = x * delta[0];

      const float mX = X * matrix[0][0] + mXo;
      const float mY = X * matrix[0][1] + mYo;
      const float mZ = X * matrix[0][2] + mZo;

      const float data = tex3D( texRef, x, y, z );
      const float dataX = tex3D( texRefX, mX, mY, mZ );
      
      const float diff = data-dataX;
      sq[tx] += diff*diff;
    }

  // compute sum via butterfly
  for ( int bit = 1; bit < blockDim.x; bit <<= 1 )
    {
      const float sum = sq[tx] + sq[tx^bit];
      __syncthreads();
      sq[tx] = sum;
      __syncthreads();
    }
}

float
cmtkImageSymmetryPlaneFunctionalDeviceEvaluate( const int* dims3, void* array, const float matrix[4][4], const float delta[3] )
{
  // Set texture parameters
  texRef.addressMode[0] = hipAddressModeWrap;
  texRef.addressMode[1] = hipAddressModeWrap;
  texRef.addressMode[2] = hipAddressModeWrap;
  texRef.filterMode = hipFilterModeLinear; 
  texRef.normalized = true; 

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc( 32, 0, 0, 0, hipChannelFormatKindFloat );
  
  // Bind the array to the texture reference 
  hipBindTextureToArray( texRef, (struct hipArray*) array, channelDesc );

  // Set texture parameters
  texRefX.addressMode[0] = hipAddressModeClamp;
  texRefX.addressMode[1] = hipAddressModeClamp;
  texRefX.addressMode[2] = hipAddressModeClamp;
  texRefX.filterMode = hipFilterModePoint; 
  texRefX.normalized = false; 

  hipBindTextureToArray( texRefX, (struct hipArray*) array, channelDesc );

  dim3 dimBlock( 32, dims3[1], 1 );
  dim3 dimGrid( dims3[2], 1 );
  
  cmtkImageSymmetryPlaneFunctionalDeviceEvaluateKernel<<<dimGrid,dimBlock>>>( matrix, delta, dims3[0], dims3[1], dims3[2] );

  const hipError_t kernelError = hipGetLastError();
  if ( kernelError != hipSuccess )
    {
      fprintf( stderr, "ERROR: CUDA kernel failed with error %s\n",hipGetErrorString( kernelError ) );
      exit( 1 );      
    }

  hipUnbindTexture( texRef );

  return 0;
}
