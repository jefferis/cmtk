/*
//
//  Copyright 2010 SRI International
//
//  This file is part of the Computational Morphometry Toolkit.
//
//  http://www.nitrc.org/projects/cmtk/
//
//  The Computational Morphometry Toolkit is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  The Computational Morphometry Toolkit is distributed in the hope that it
//  will be useful, but WITHOUT ANY WARRANTY; without even the implied
//  warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with the Computational Morphometry Toolkit.  If not, see
//  <http://www.gnu.org/licenses/>.
//
//  $Revision$
//
//  $LastChangedDate$
//
//  $LastChangedBy$
//
*/

#include "cmtkImageSymmetryPlaneFunctionalDevice_kernels.h"

#include <hip/hip_runtime_api.h>

texture<float, 3, hipReadModeElementType> texRef;

float
cmtkImageSymmetryPlaneFunctionalDeviceEvaluate( void* array )
{
  // Set texture parameters
  texRef.addressMode[0] = hipAddressModeWrap; 
  texRef.addressMode[1] = hipAddressModeWrap; 
  texRef.filterMode = hipFilterModeLinear; 
  texRef.normalized = false; 

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  
  // Bind the array to the texture reference 
  hipBindTextureToArray( texRef, (struct hipArray*) array, channelDesc );

  return 0;
}
