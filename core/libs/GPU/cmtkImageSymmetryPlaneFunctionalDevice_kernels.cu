/*
//
//  Copyright 2010 SRI International
//
//  This file is part of the Computational Morphometry Toolkit.
//
//  http://www.nitrc.org/projects/cmtk/
//
//  The Computational Morphometry Toolkit is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  The Computational Morphometry Toolkit is distributed in the hope that it
//  will be useful, but WITHOUT ANY WARRANTY; without even the implied
//  warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with the Computational Morphometry Toolkit.  If not, see
//  <http://www.gnu.org/licenses/>.
//
//  $Revision$
//
//  $LastChangedDate$
//
//  $LastChangedBy$
//
*/

#include "cmtkImageSymmetryPlaneFunctionalDevice_kernels.h"

#include <hip/hip_runtime_api.h>

#include <cstdio>

/// Texture reference to volume data.
texture<float, 3, hipReadModeElementType> texRef;

__constant__ float deviceAxesTNL[16384];

float
cmtkImageSymmetryPlaneFunctionalDeviceEvaluate( const int* dims3, void* array, const float* axesTNL )
{
  if ( (hipMemcpyToSymbol(HIP_SYMBOL( deviceAxesTNL), axesTNL, 3*(dims3[0]+dims3[1]+dims3[2])*sizeof( *axesTNL ), 0, hipMemcpyHostToDevice ) != hipSuccess) )
    {
      fprintf( stderr, "ERROR: hipMemcpy() to constant memory failed with error %s\n",hipGetErrorString( hipGetLastError() ) );
      exit( 1 );      
    }
  
  // Set texture parameters
  texRef.addressMode[0] = hipAddressModeWrap;
  texRef.addressMode[1] = hipAddressModeWrap;
  texRef.addressMode[2] = hipAddressModeWrap;
  texRef.filterMode = hipFilterModeLinear; 
  texRef.normalized = true; 

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  
  // Bind the array to the texture reference 
  hipBindTextureToArray( texRef, (struct hipArray*) array, channelDesc );

  hipUnbindTexture( texRef );

  return 0;
}
