#include "hip/hip_runtime.h"
/*
//
//  Copyright 2010 SRI International
//
//  This file is part of the Computational Morphometry Toolkit.
//
//  http://www.nitrc.org/projects/cmtk/
//
//  The Computational Morphometry Toolkit is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  The Computational Morphometry Toolkit is distributed in the hope that it
//  will be useful, but WITHOUT ANY WARRANTY; without even the implied
//  warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with the Computational Morphometry Toolkit.  If not, see
//  <http://www.gnu.org/licenses/>.
//
//  $Revision$
//
//  $LastChangedDate$
//
//  $LastChangedBy$
//
*/

#include "cmtkImageSymmetryPlaneFunctionalDevice_kernels.h"

#include <hip/hip_runtime_api.h>

#include <cstdio>

/// Texture reference to volume data.
texture<float, 3, hipReadModeElementType> texRef;
texture<float, 3, hipReadModeElementType> texRefX;

__constant__ float deviceAxesTNL[16384];

__global__
void cmtkImageSymmetryPlaneFunctionalDeviceEvaluateKernel( const int dims0, const int dims1, const int dims2 )
{
  const int x = threadIdx.x;
  const int y = threadIdx.y;
  const int z = blockIdx.x;

  const int offsetX = 3 * x;
  const int offsetY = 3 * (dims0 + y);
  const int offsetZ = 3 * (dims0 + dims1 + z);

  const float xX = deviceAxesTNL[offsetX] + deviceAxesTNL[offsetY] + deviceAxesTNL[offsetZ];
  const float yX = deviceAxesTNL[offsetX+1] + deviceAxesTNL[offsetY+1] + deviceAxesTNL[offsetZ+1];
  const float zX = deviceAxesTNL[offsetX+2] + deviceAxesTNL[offsetY+2] + deviceAxesTNL[offsetZ+2];

  const float data = tex3D( texRef, x, y, z );
  const float dataX = tex3D( texRefX, xX, yX, zX );

  const float diff = data-dataX;
  const float sq = diff*diff;
}

float
cmtkImageSymmetryPlaneFunctionalDeviceEvaluate( const int* dims3, void* array, const float* axesTNL )
{
  if ( (hipMemcpyToSymbol(HIP_SYMBOL( deviceAxesTNL), axesTNL, 3*(dims3[0]+dims3[1]+dims3[2])*sizeof( *axesTNL ), 0, hipMemcpyHostToDevice ) != hipSuccess) )
    {
      fprintf( stderr, "ERROR: hipMemcpy() to constant memory failed with error %s\n",hipGetErrorString( hipGetLastError() ) );
      exit( 1 );      
    }
  
  // Set texture parameters
  texRef.addressMode[0] = hipAddressModeWrap;
  texRef.addressMode[1] = hipAddressModeWrap;
  texRef.addressMode[2] = hipAddressModeWrap;
  texRef.filterMode = hipFilterModeLinear; 
  texRef.normalized = true; 

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  
  // Bind the array to the texture reference 
  hipBindTextureToArray( texRef, (struct hipArray*) array, channelDesc );

  // Set texture parameters
  texRefX.addressMode[0] = hipAddressModeClamp;
  texRefX.addressMode[1] = hipAddressModeClamp;
  texRefX.addressMode[2] = hipAddressModeClamp;
  texRefX.filterMode = hipFilterModePoint; 
  texRefX.normalized = false; 

  hipBindTextureToArray( texRefX, (struct hipArray*) array, channelDesc );

  dim3 dimBlock( dims3[0], dims3[1], 1 );
  dim3 dimGrid( dims3[2], 1 );
  
  cmtkImageSymmetryPlaneFunctionalDeviceEvaluateKernel<<<dimGrid,dimBlock>>>( dims3[0], dims3[1], dims3[2] );

  const hipError_t kernelError = hipGetLastError();
  if ( kernelError != hipSuccess )
    {
      fprintf( stderr, "ERROR: CUDA kernel failed with error %s\n",hipGetErrorString( kernelError ) );
      exit( 1 );      
    }

  hipUnbindTexture( texRef );

  return 0;
}
