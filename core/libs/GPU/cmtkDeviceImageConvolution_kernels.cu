#include "hip/hip_runtime.h"
/*
//
//  Copyright 2010 SRI International
//
//  This file is part of the Computational Morphometry Toolkit.
//
//  http://www.nitrc.org/projects/cmtk/
//
//  The Computational Morphometry Toolkit is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  The Computational Morphometry Toolkit is distributed in the hope that it
//  will be useful, but WITHOUT ANY WARRANTY; without even the implied
//  warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with the Computational Morphometry Toolkit.  If not, see
//  <http://www.gnu.org/licenses/>.
//
//  $Revision$
//
//  $LastChangedDate$
//
//  $LastChangedBy$
//
*/

#include "cmtkDeviceImageConvolution_kernels.h"

#include "System/cmtkMemory.h"

#include "GPU/cmtkCUDA.h"
#include "GPU/cmtkDeviceMemory.h"

#include <hip/hip_runtime_api.h>

/// Texture reference to volume data.
texture<float, 3, hipReadModeElementType> texRef;

__constant__ float deviceKernel[128];

__global__
void
cmtkDeviceImageConvolutionKernelX( float* dest, int dims0, int dims1, int dims2, int kernelLength )
{
}

__global__
void
cmtkDeviceImageConvolutionKernelY( float* dest, int dims0, int dims1, int dims2, int kernelLength )
{
}

__global__
void
cmtkDeviceImageConvolutionKernelZ( float* dest, int dims0, int dims1, int dims2, int kernelLength )
{
}

void
cmtkDeviceImageConvolution( const int* dims3, void* array, const int kernelLengthX, const float* kernelX, const int kernelLengthY, const float* kernelY, const int kernelLengthZ, const float* kernelZ )
{
  // Set texture parameters for fixed image indexed access
  texRef.addressMode[0] = hipAddressModeClamp;
  texRef.addressMode[1] = hipAddressModeClamp;
  texRef.addressMode[2] = hipAddressModeClamp;
  texRef.filterMode = hipFilterModePoint; 
  texRef.normalized = false; 

  cmtkCheckCallCUDA( hipBindTextureToArray( texRef, (struct hipArray*) array, hipCreateChannelDesc<float>() ) );

  const int nPixels = dims3[0] * dims3[1] * dims3[2];
  cmtk::DeviceMemory<float>::SmartPtr temporary = cmtk::DeviceMemory<float>::Create( nPixels );

  cmtkCheckCallCUDA( hipMemcpyToSymbol(HIP_SYMBOL( deviceKernel), kernelX, kernelLengthX * sizeof( float ), 0, hipMemcpyHostToDevice ) );
  
  dim3 threads;
  dim3 blocks;

  cmtkDeviceImageConvolutionKernelX<<<threads,blocks>>>( temporary->Ptr(), dims3[0], dims3[1], dims3[2], kernelLengthX );
  cmtkCheckLastErrorCUDA;

  cmtkCheckCallCUDA( hipMemcpyToArray( (struct hipArray*) array, 0, 0, temporary->Ptr(), nPixels, hipMemcpyDeviceToDevice ) );

  cmtkCheckCallCUDA( hipMemcpyToSymbol(HIP_SYMBOL( deviceKernel), kernelY, kernelLengthY * sizeof( float ), 0, hipMemcpyHostToDevice ) );
  cmtkDeviceImageConvolutionKernelY<<<threads,blocks>>>( temporary->Ptr(), dims3[0], dims3[1], dims3[2], kernelLengthY );
  cmtkCheckLastErrorCUDA;

  cmtkCheckCallCUDA( hipMemcpyToArray( (struct hipArray*) array, 0, 0, temporary->Ptr(), nPixels, hipMemcpyDeviceToDevice ) );

  cmtkCheckCallCUDA( hipMemcpyToSymbol(HIP_SYMBOL( deviceKernel), kernelZ, kernelLengthZ * sizeof( float ), 0, hipMemcpyHostToDevice ) );  
  cmtkDeviceImageConvolutionKernelZ<<<threads,blocks>>>( temporary->Ptr(), dims3[0], dims3[1], dims3[2], kernelLengthZ );
  cmtkCheckLastErrorCUDA;
  
  cmtkCheckCallCUDA( hipMemcpyToArray( (struct hipArray*) array, 0, 0, temporary->Ptr(), nPixels, hipMemcpyDeviceToDevice ) );
  
  cmtkCheckCallCUDA( hipUnbindTexture( texRef ) );
}
