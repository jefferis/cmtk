#include "hip/hip_runtime.h"
/*
//
//  Copyright 2010 SRI International
//
//  This file is part of the Computational Morphometry Toolkit.
//
//  http://www.nitrc.org/projects/cmtk/
//
//  The Computational Morphometry Toolkit is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  The Computational Morphometry Toolkit is distributed in the hope that it
//  will be useful, but WITHOUT ANY WARRANTY; without even the implied
//  warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with the Computational Morphometry Toolkit.  If not, see
//  <http://www.gnu.org/licenses/>.
//
//  $Revision$
//
//  $LastChangedDate$
//
//  $LastChangedBy$
//
*/

#include "cmtkDeviceImageConvolution_kernels.h"

#include "System/cmtkMemory.h"
#include "GPU/cmtkDeviceMemory.h"

#include <hip/hip_runtime_api.h>

#include <cstdio>

/// Texture reference to volume data.
texture<float, 3, hipReadModeElementType> texRef;

__constant__ float deviceKernel[128];

__global__
void
cmtkDeviceImageConvolutionKernelX( float* dest, int dims0, int dims1, int dims2, int kernelLength )
{
}

__global__
void
cmtkDeviceImageConvolutionKernelY( float* dest, int dims0, int dims1, int dims2, int kernelLength )
{
}

__global__
void
cmtkDeviceImageConvolutionKernelZ( float* dest, int dims0, int dims1, int dims2, int kernelLength )
{
}

void
cmtkDeviceImageConvolution( const int* dims3, void* array, const int kernelLengthX, const float* kernelX, const int kernelLengthY, const float* kernelY, const int kernelLengthZ, const float* kernelZ )
{
  // Set texture parameters for fixed image indexed access
  texRef.addressMode[0] = hipAddressModeClamp;
  texRef.addressMode[1] = hipAddressModeClamp;
  texRef.addressMode[2] = hipAddressModeClamp;
  texRef.filterMode = hipFilterModePoint; 
  texRef.normalized = false; 

  hipError_t hipError_t = hipBindTextureToArray( texRef, (struct hipArray*) array, hipCreateChannelDesc<float>() );
  if ( hipError_t != hipSuccess )
    {
      fprintf( stderr, "ERROR: hipBindTextureToArray failed with error '%s'\n", hipGetErrorString( hipError_t ) );
      exit( 1 );      
    }

  const int nPixels = dims3[0] * dims3[1] * dims3[2];
  cmtk::DeviceMemory<float>::SmartPtr temporary = cmtk::DeviceMemory<float>::Create( nPixels );

  if ( (hipMemcpyToSymbol(HIP_SYMBOL( deviceKernel), kernelX, kernelLengthX * sizeof( float ), 0, hipMemcpyHostToDevice ) != hipSuccess) )
    {
      fprintf( stderr, "ERROR: hipMemcpyToSymbol() to constant memory failed with error %s\n",hipGetErrorString( hipGetLastError() ) );
      exit( 1 );      
    }
  
  dim3 threads;
  dim3 blocks;

  cmtkDeviceImageConvolutionKernelX<<<threads,blocks>>>( temporary->Ptr(), dims3[0], dims3[1], dims3[2], kernelLengthX );

  hipMemcpyToArray( (struct hipArray*) array, 0, 0, temporary->Ptr(), nPixels, hipMemcpyDeviceToDevice );

  if ( (hipMemcpyToSymbol(HIP_SYMBOL( deviceKernel), kernelY, kernelLengthY * sizeof( float ), 0, hipMemcpyHostToDevice ) != hipSuccess) )
    {
      fprintf( stderr, "ERROR: hipMemcpyToSymbol() to constant memory failed with error %s\n",hipGetErrorString( hipGetLastError() ) );
      exit( 1 );      
    }
  
  cmtkDeviceImageConvolutionKernelY<<<threads,blocks>>>( temporary->Ptr(), dims3[0], dims3[1], dims3[2], kernelLengthY );

  hipMemcpyToArray( (struct hipArray*) array, 0, 0, temporary->Ptr(), nPixels, hipMemcpyDeviceToDevice );

  if ( (hipMemcpyToSymbol(HIP_SYMBOL( deviceKernel), kernelZ, kernelLengthZ * sizeof( float ), 0, hipMemcpyHostToDevice ) != hipSuccess) )
    {
      fprintf( stderr, "ERROR: hipMemcpyToSymbol() to constant memory failed with error %s\n",hipGetErrorString( hipGetLastError() ) );
      exit( 1 );      
    }
  
  cmtkDeviceImageConvolutionKernelZ<<<threads,blocks>>>( temporary->Ptr(), dims3[0], dims3[1], dims3[2], kernelLengthZ );

  hipMemcpyToArray( (struct hipArray*) array, 0, 0, temporary->Ptr(), nPixels, hipMemcpyDeviceToDevice );
  
  hipUnbindTexture( texRef );
}
